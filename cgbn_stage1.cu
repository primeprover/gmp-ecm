#include "hip/hip_runtime.h"
/***

Copyright (c) 2018-2019, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.

***/

#include "cgbn_stage1.h"

#include <stdio.h>

#include <cgbn.h>
#include <hip/hip_runtime.h>

void cuda_check(hipError_t status, const char *action=NULL, const char *file=NULL, int32_t line=0) {
  // check for cuda errors

  if(status!=hipSuccess) {
    printf("CUDA error occurred: %s\n", hipGetErrorString(status));
    if(action!=NULL)
      printf("While running %s   (file %s, line %d)\n", action, file, line);
    exit(1);
  }
}

// support routine copied from  "CGBN/samples/utility/support.h"
void cgbn_check(cgbn_error_report_t *report, const char *file=NULL, int32_t line=0) {
  // check for cgbn errors

  if(cgbn_error_report_check(report)) {
    printf("\n");
    printf("CGBN error occurred: %s\n", cgbn_error_string(report));

    if(report->_instance!=0xFFFFFFFF) {
      printf("Error reported by instance %d", report->_instance);
      if(report->_blockIdx.x!=0xFFFFFFFF || report->_threadIdx.x!=0xFFFFFFFF)
        printf(", ");
      if(report->_blockIdx.x!=0xFFFFFFFF)
      printf("blockIdx=(%d, %d, %d) ", report->_blockIdx.x, report->_blockIdx.y, report->_blockIdx.z);
      if(report->_threadIdx.x!=0xFFFFFFFF)
        printf("threadIdx=(%d, %d, %d)", report->_threadIdx.x, report->_threadIdx.y, report->_threadIdx.z);
      printf("\n");
    }
    else {
      printf("Error reported by blockIdx=(%d %d %d)", report->_blockIdx.x, report->_blockIdx.y, report->_blockIdx.z);
      printf("threadIdx=(%d %d %d)\n", report->_threadIdx.x, report->_threadIdx.y, report->_threadIdx.z);
    }
    if(file!=NULL)
      printf("file %s, line %d\n", file, line);
    exit(1);
  }
}

// Unify this with cudakernel.cu
#define CUDA_CHECK(action) cuda_check(action, #action, __FILE__, __LINE__)
#define CGBN_CHECK(report) cgbn_check(report, __FILE__, __LINE__)

void to_mpz(mpz_t r, const uint32_t *x, uint32_t count) {
  mpz_import(r, count, -1, sizeof(uint32_t), 0, 0, x);
}

void from_mpz(const mpz_t s, uint32_t *x, uint32_t count) {
  size_t words;

  if(mpz_sizeinbase(s, 2)>count*32) {
    fprintf(stderr, "from_mpz failed -- result does not fit\n");
    exit(1);
  }

  mpz_export(x, &words, -1, sizeof(uint32_t), 0, 0, s);
  while(words<count)
    x[words++]=0;
}


// ---------------------------------------------------------------- //

// IMPORTANT:  DO NOT DEFINE TPI OR BITS BEFORE INCLUDING CGBN
#define TPI 32
#define BITS 1024

// Declare the instance type
typedef struct {
  cgbn_mem_t<BITS> a;
  cgbn_mem_t<BITS> b;
  cgbn_mem_t<BITS> sum;
} instance_t;


// helpful typedefs for the kernel
typedef cgbn_context_t<TPI>         context_t;
typedef cgbn_env_t<context_t, BITS> env_t;

// the actual kernel
__global__ void kernel_add(cgbn_error_report_t *report, instance_t *instances, uint32_t count) {
  int32_t instance;

  // decode an instance number from the blockIdx and threadIdx
  instance=(blockIdx.x*blockDim.x + threadIdx.x)/TPI;
  if(instance>=count)
    return;

  context_t      bn_context(cgbn_report_monitor, report, instance);   // construct a context
  env_t          bn_env(bn_context.env<env_t>());                     // construct an environment for 1024-bit math
  env_t::cgbn_t  a, b, r;                                             // define a, b, r as 1024-bit bignums

  cgbn_load(bn_env, a, &(instances[instance].a));      // load my instance's a value
  cgbn_load(bn_env, b, &(instances[instance].b));      // load my instance's b value
  cgbn_add(bn_env, r, a, b);                           // r=a+b
  cgbn_store(bn_env, &(instances[instance].sum), r);   // store r into sum
}

extern "C"
void run_cgbn(mpz_t N, mpz_t s) {
  instance_t          *instances, *gpuInstances;
  cgbn_error_report_t *report;

  instances = (instance_t *) malloc(sizeof(instance_t));
  from_mpz(N, instances[0].a._limbs, BITS/32);
  from_mpz(N, instances[0].b._limbs, BITS/32);

  printf("Copying instances to the GPU ...\n");
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(instance_t)));
  CUDA_CHECK(hipMemcpy(gpuInstances, instances, sizeof(instance_t), hipMemcpyHostToDevice));

  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  printf("Running GPU kernel ...\n");
  // launch with 32 threads per instance, 128 threads (4 instances) per block
  kernel_add<<<(1+3)/4, 128>>>(report, gpuInstances, 1);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);

  // copy the instances back from gpuMemory
  printf("Copying results back to CPU ...\n");
  CUDA_CHECK(hipMemcpy(instances, gpuInstances, sizeof(instance_t), hipMemcpyDeviceToHost));

  printf("Verifying the results ...\n");
  mpz_t result;
  mpz_init(result);
  to_mpz(result, instances[0].sum._limbs, BITS/32);
  gmp_printf("2*N(%Zd) = %Zd\n", N, result);
  mpz_sub(result, result, N);
  mpz_sub(result, result, N);
  printf("%sVALID!\n", mpz_cmp_ui(result, 0) == 0 ? "" : "IN");
  mpz_clear(result);

  // clean up
  //free(instances);
  CUDA_CHECK(hipFree(gpuInstances));
  CUDA_CHECK(cgbn_error_report_free(report));
}
