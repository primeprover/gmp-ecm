#include "hip/hip_runtime.h"
#include "def.h"
#include "cudakernel.h"

__constant__ __device__ digit_t d_invNcst;
__device__ biguint_t d_Ncst;
__device__ biguint_t d_3Ncst;
__device__ biguint_t d_Mcst;


#define errCheck(err) cuda_errCheck (err, __FILE__, __LINE__)
#define hipMalloc(d, size) errCheck (hipMalloc (d, size))
#define cudaMemcpyHtoD(d, h, size) errCheck (hipMemcpy ((void *) d, \
                                    (void *) h, size, hipMemcpyHostToDevice))
#define cudaMemcpyDtoH(h, d, size) errCheck (hipMemcpy ((void *) h, \
                                    (void *) d, size, hipMemcpyDeviceToHost))


/******************************/
/* Host code handling the GPU */
/******************************/

inline void cuda_errCheck (hipError_t err, const char *file, const int line)
{
  if( err != hipSuccess ) 
  {
    fprintf(stderr, "%s(%i) : Error cuda : %s.\n",
              file, line, hipGetErrorString( err) );
    exit(EXIT_FAILURE);
  }
}

/* First call to a global function initialize the device */
__global__ void Cuda_Init_Device ()
{
}

extern "C" 
int select_and_init_GPU (int device, int number_of_curves, FILE *OUTPUT_VERBOSE)
{
  hipDeviceProp_t deviceProp;
  hipError_t err;
        
  fprintf(OUTPUT_VERBOSE, "#Compiled for a NVIDIA GPU with " 
          "compute capability %d.%d.\n", MAJOR, MINOR);

  if (device!=-1)
  {
    fprintf(OUTPUT_VERBOSE,"#Device %d is required.\n",device);

    err= hipSetDevice(device);
    if (err != hipSuccess)
    {
      fprintf(stderr, "Error: Could not use device %d\n",device);
      fprintf(stderr, "Error msg: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  }
  
  err = hipGetDevice (&device);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Error: no active device\n");
    fprintf(stderr, "Error msg: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipGetDeviceProperties (&deviceProp, device);
  if (err != hipSuccess)
  {
    fprintf(stderr, "Error while getting device's properties\n");
    fprintf(stderr, "Error msg: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  int minor = deviceProp.minor;
  int major = deviceProp.major;
  int MPcount = deviceProp.multiProcessorCount;

  if (10 * major + minor < 10 * MAJOR + MINOR)
  {
    fprintf(stderr, "Error: Device %d have a compute capability of %d.%d " 
                    "(required %d.%d).\n", device, major, minor, MAJOR, MINOR);
    exit(EXIT_FAILURE);
  }

  fprintf(OUTPUT_VERBOSE, "#Will use device %d : %s, compute capability %d.%d, "
          "%d MPs.\n", device, deviceProp.name, major, minor, MPcount);


  hipSetDeviceFlags(hipDeviceScheduleAuto); 
  //hipSetDeviceFlags(hipDeviceScheduleYield); 
  //hipSetDeviceFlags(hipDeviceScheduleSpin); //the other make performance
  //worse

  /* number_of_curves should be a multiple of CURVES_BY_BLOCK */
  number_of_curves=(number_of_curves/CURVES_BY_BLOCK)*CURVES_BY_BLOCK;
  if (number_of_curves==0)
    number_of_curves = MPcount * CURVES_BY_MP;

  /* First call to a global function initialize the device */
  Cuda_Init_Device<<<1, 1>>> ();
  errCheck (hipGetLastError()); 

  return number_of_curves;
}

extern "C"
void cuda_Main (biguint_t h_N, biguint_t h_3N, biguint_t h_M, digit_t h_invN, 
                    biguint_t *h_xarray, biguint_t *h_zarray, 
                    biguint_t *h_x2array, biguint_t *h_z2array, mpz_t s,
                    unsigned int firstinvd, unsigned int number_of_curves, 
                    FILE *OUTPUT_VERBOSE, FILE *OUTPUT_VVERBOSE) 
{ 
  size_t j;
  biguint_t *d_xA, *d_zA, *d_xB, *d_zB;

  size_t array_size = sizeof(biguint_t) * number_of_curves;

  dim3 dimBlock (NB_DIGITS, CURVES_BY_BLOCK);
  dim3 dimGrid (number_of_curves/CURVES_BY_BLOCK);

  fprintf(OUTPUT_VVERBOSE, "Block: %ux%ux%u Grid: %ux%ux%u\n", dimBlock.x, 
                      dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);

  hipMalloc (&d_xA, array_size);
  hipMalloc (&d_zA, array_size);
  hipMalloc (&d_xB, array_size);
  hipMalloc (&d_zB, array_size);

  /* Copy into the gpu memory */
  hipMemcpyToSymbol(HIP_SYMBOL(d_invNcst), (void *) &h_invN, sizeof(digit_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_Ncst), (void *) h_N, sizeof(biguint_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_3Ncst), (void *) h_3N, sizeof(biguint_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_Mcst), (void *) h_M, sizeof(biguint_t));

  cudaMemcpyHtoD (d_xA, h_xarray, array_size);
  cudaMemcpyHtoD (d_zA, h_zarray, array_size);
  cudaMemcpyHtoD (d_xB, h_x2array, array_size);
  cudaMemcpyHtoD (d_zB, h_z2array, array_size);

  /* Double-and-add loop: it calls the GPU for each bits of s */
  for (j = mpz_sizeinbase (s, 2) - 1; j>0; j-- )
  {
    if (mpz_tstbit (s, j-1) == 1)
      Cuda_Ell_DblAdd<<<dimGrid,dimBlock>>>(d_xB, d_zB, d_xA, d_zA, firstinvd);
    else
      Cuda_Ell_DblAdd<<<dimGrid,dimBlock>>>(d_xA, d_zA, d_xB, d_zB, firstinvd);

    //maybe only for debug mode??
    errCheck (hipGetLastError()); 
  }

  /* Get the results back from device memory */
  cudaMemcpyDtoH (h_xarray, d_xA, array_size);
  cudaMemcpyDtoH (h_zarray, d_zA, array_size);

  hipFree ((void *) d_xA);
  hipFree ((void *) d_zA);
  hipFree ((void *) d_xB);
  hipFree ((void *) d_zB);
}



/***************/
/* Device code */
/***************/


#define __add_cc(r,a,b) asm ("add.cc.u32 %0, %1, %2;": "=r"(r): "r"(a), "r"(b)) 
#define __addc_cc(r,a,b) asm ("addc.cc.u32 %0, %1, %2;": "=r"(r): "r"(a), "r"(b))
#define __sub_cc(r,a,b) asm ("sub.cc.u32 %0, %1, %2;": "=r"(r): "r"(a), "r"(b)) 

#define __addcy(carry) asm ("addc.s32 %0, 0, 0;" : "=r"(carry)) 
#define __addcy2(carry) asm ("addc.s32 %0, %0, 0;" : "+r"(carry)) 

#define __subcy(carry) asm ("subc.s32 %0, 0, 0;" : "=r"(carry)) 
#define __subcy2(carry) asm ("subc.s32 %0, %0, 0;" : "+r"(carry)) 

#define __mul(h, l, a, b) asm("mul.hi.u32 %0, %2, %3;\n\t"\
                              "mul.lo.u32 %1, %2, %3;"\
                                   : "=r"(h), "=r"(l) : "r"(a), "r"(b))


/* Assume cy[threadIdx.x] = 0,+/-1 */
__device__ void Cuda_Normalize (biguint_t A, dbigint_t cy)
{
  carry_t cytemp;
  int tmp = (threadIdx.x - 1) % NB_DIGITS;
  cytemp = cy[tmp];

  __add_cc(A[threadIdx.x], A[threadIdx.x], cytemp);
  
  if (cytemp >= 0)
    __addcy(cy[threadIdx.x]);
  else /* if (cytemp < 0) */
    __subcy(cy[threadIdx.x]);
}

__device__ void Cuda_Fully_Normalize (biguint_t A, dbigint_t cy)
{
  do
  {
    Cuda_Normalize(A,cy);
  }while(__any(cy[threadIdx.x])!=0);
}

/* Compute Rmod <- A + B */ 
/* Input: 0 <= A, B < 3*N */ 
/* Ouput: 0 <= Rmod < 6*N */ 
__device__ void Cuda_Add_mod
(biguint_t Rmod, dbigint_t cy, const biguint_t A, const biguint_t B)
{
  __add_cc (Rmod[threadIdx.x], A[threadIdx.x], B[threadIdx.x]);
  __addcy (cy[threadIdx.x]);
  Cuda_Fully_Normalize (Rmod, cy); 
}

/* Compute Rmod <- Rmod + B */ 
/* Input: 0 <= Rmod, B < 3*N */ 
/* (except when it follows Cuda_Mulint_mod, 0 <= Rmod < 3*N, 0 < B < 7*N ) */ 
/* Ouput: 0 <= Rmod < 6*N */ 
/* (except when it follows Cuda_Mulint_mod, 0 <= Rmod < 10*N) */ 
__device__ void Cuda_Add_mod
(biguint_t Rmod, dbigint_t cy, const biguint_t A)
{
  __add_cc (Rmod[threadIdx.x], Rmod[threadIdx.x], A[threadIdx.x]);
  __addcy (cy[threadIdx.x]);
  Cuda_Fully_Normalize (Rmod, cy);
}

/* Compute Rmod <- Rmod - B */ 
/* Input: 0 <= Rmod, B < 3*N */ 
/* Ouput: 0 <= Rmod < 6*N */ 
__device__ void Cuda_Sub_mod 
(biguint_t Rmod, dbigint_t cy, const biguint_t B, const digit_t N3thdx)
{
  __add_cc (Rmod[threadIdx.x], Rmod[threadIdx.x], N3thdx);
  __addcy (cy[threadIdx.x]);
  __sub_cc (Rmod[threadIdx.x], Rmod[threadIdx.x], B[threadIdx.x]);
  __subcy2 (cy[threadIdx.x]);
  Cuda_Fully_Normalize (Rmod, cy); 
}

/* Perform one step of REDC */ 
__device__ void Cuda_Mulmod_step
(dbiguint_t r, dbigint_t cy, digit_t a, digit_t b, const digit_t Nthdx,
 const digit_t invN)
#ifdef CC13
{
  digit_t h,l;
  int tmp;
  __mul(h,l,a,b);
  __add_cc(r[threadIdx.x],r[threadIdx.x],l);
  __addc_cc(r[threadIdx.x+1],r[threadIdx.x+1],h);
  __addcy2(cy[threadIdx.x+1]);


  __mul(h, l, invN*r[0], Nthdx);
  __add_cc(r[threadIdx.x],r[threadIdx.x],l);
  __addc_cc(r[threadIdx.x+1],r[threadIdx.x+1],h);
  __addcy2(cy[threadIdx.x+1]);
 
  //make one round of normalize + a right shift
  __add_cc(r[threadIdx.x],r[threadIdx.x+1],cy[threadIdx.x]);
  tmp=(threadIdx.x==NB_DIGITS-1)?cy[threadIdx.x+1]:0;
  __asm__("addc.s32 %0,%1, 0;" :"=r"(cy[threadIdx.x]): "r"(tmp)); 

  if (threadIdx.x==0)
  {
    cy[NB_DIGITS]=0;
    r[NB_DIGITS]=0;
  }
}
#else /* Use madc which exists only on CC 2.x cards */
{
  digit_t t;
  int tmp;
  asm ("mad.lo.cc.u32 %0, %1, %2, %0;" : "+r"(r[threadIdx.x]) :
                                                  "r"(a), "r"(b));
  asm ("madc.hi.cc.u32 %0, %1, %2, %0;" : "+r"(r[threadIdx.x+1]) :
                                                  "r"(a), "r"(b));
  __addcy2(cy[threadIdx.x+1]);

  asm ("mul.lo.u32 %0, %1, %2;" : "=r"(t) : "r"(invN), "r"(r[0]));
  asm ("mad.lo.cc.u32 %0, %1, %2, %0;" : "+r"(r[threadIdx.x]) :
                                              "r"(t), "r"(Nthdx));
  asm ("madc.hi.cc.u32 %0, %1, %2, %0;" : "+r"(r[threadIdx.x+1]) : 
                                              "r"(t), "r"(Nthdx));
  __addcy2(cy[threadIdx.x+1]);

  //make one round of normalize + a right shift
  __add_cc(r[threadIdx.x],r[threadIdx.x+1],cy[threadIdx.x]);
  tmp=(threadIdx.x==NB_DIGITS-1)?cy[threadIdx.x+1]:0;
  __asm__("addc.s32 %0,%1, 0;" :"=r"(cy[threadIdx.x]): "r"(tmp)); 

  if (threadIdx.x==0)
  {
    cy[NB_DIGITS]=0;
    r[NB_DIGITS]=0;
  }
}
#endif

/* Compute r <- 2*a */ 
/* Input: 0 <= a < 3*N */ 
/* Ouput: 0 <= r < 3*N */ 
__device__ void Cuda_Dbl_mod
(biguint_t r, biguint_t a)
{
  asm ("add.cc.u32 %0, %1, %1;" : "=r"(r[threadIdx.x]) : "r"(a[threadIdx.x]));
  __addcy2(r[threadIdx.x+1]);
}


/* Compute r <- A*b */ 
/* Input: 0 < b < 2^SIZE_DIGIT, 0 <= A < 6*N */ 
/* Ouput: 0 <= r < 7*N */ 
__device__ void Cuda_Mulint_mod
(dbiguint_t r, dbigint_t cy, biguint_t A, digit_t b, const digit_t Nthdx,
 const digit_t invN)
#ifdef CC13
{
  digit_t h,l;
  __mul(h, r[threadIdx.x], A[threadIdx.x], b);
  __add_cc(r[threadIdx.x+1], r[threadIdx.x+1], h);
  __addcy(cy[threadIdx.x+1]);

  //h*2^32+l =A[i]*B[threadIDx.x]
  __mul(h, l, invN*r[0], Nthdx);
  __add_cc(r[threadIdx.x], r[threadIdx.x],l);
  __addc_cc(r[threadIdx.x+1], r[threadIdx.x+1],h);
  __addcy2(cy[threadIdx.x+1]);

  __add_cc(r[threadIdx.x], r[threadIdx.x+1], cy[threadIdx.x]);
  __addcy(cy[threadIdx.x]);

  if (threadIdx.x==0)
    r[NB_DIGITS]=0;
  
  Cuda_Fully_Normalize(r,cy); 
}
#else /* Use madc which exists only on CC 2.x cards */
{
  digit_t t;
  asm ("mul.lo.u32 %0, %1, %2;" : "=r"(r[threadIdx.x]) :
                                                  "r"(A[threadIdx.x]), "r"(b));
  asm ("mad.hi.cc.u32 %0, %1, %2, %0;" : "+r"(r[threadIdx.x+1]) :
                                                  "r"(A[threadIdx.x]), "r"(b));
  __addcy(cy[threadIdx.x+1]);

  asm ("mul.lo.u32 %0, %1, %2;" : "=r"(t) : "r"(invN), "r"(r[0]));
  asm ("mad.lo.cc.u32 %0, %1, %2, %0;" : "+r"(r[threadIdx.x]) :
                                              "r"(t), "r"(Nthdx));
  asm ("madc.hi.cc.u32 %0, %1, %2, %0;" : "+r"(r[threadIdx.x+1]) :
                                              "r"(t), "r"(Nthdx));
  __addcy2(cy[threadIdx.x+1]);

  __add_cc(r[threadIdx.x], r[threadIdx.x+1], cy[threadIdx.x]);
  __addcy(cy[threadIdx.x]);

  if (threadIdx.x==0)
    r[NB_DIGITS]=0;
  
  Cuda_Fully_Normalize(r,cy); 
}
#endif

/* Compute r <- A*B */ 
/* Input: 0 <= A, B < 6*N */
/* (except when it follows Cuda_Mulint_mod, 0 <= A < 6*N, 0 < B < 10*N ) */ 
/* Ouput: 0 <= r < 3*N */ 
__device__ void Cuda_Mul_mod 
(biguint_t mul, dbigint_t cy, const biguint_t A, const biguint_t B, dbiguint_t r,
 const digit_t Nthdx, const digit_t invN)
{

  int i;
  digit_t temp=A[threadIdx.x];

  r[threadIdx.x]=0;
  
  for (i=0; i<NB_DIGITS; i++)
    Cuda_Mulmod_step (r, cy, temp, B[i], Nthdx, invN);

  
  Cuda_Fully_Normalize (r, cy);
  mul[threadIdx.x]=r[threadIdx.x];
}

__device__ void Cuda_Square_mod 
(biguint_t mul, dbigint_t cy, const biguint_t A, dbiguint_t r, 
 const digit_t Nthdx, const digit_t invN)
{
  Cuda_Mul_mod (mul, cy, A, A, r, Nthdx, invN);
}

/* 
  Compute silmutaneously:
  (xarg : zarg ) <- [2](xarg : zarg) 
  (xarg2 : zarg2 ) <- (xarg : zarg) + (xarg2 : zarg2) 
*/
__global__ void 
Cuda_Ell_DblAdd (biguint_t *xarg, biguint_t *zarg, biguint_t *x2arg, 
                                       biguint_t *z2arg, unsigned int firstinvd)
{
  __shared__ VOL digit_t b_temp_r[CURVES_BY_BLOCK][NB_DIGITS+1];
  __shared__ VOL carry_t b_cy[CURVES_BY_BLOCK][NB_DIGITS+1]; 

  __shared__ VOL digit_t b_t[CURVES_BY_BLOCK][NB_DIGITS];
  __shared__ VOL digit_t b_u[CURVES_BY_BLOCK][NB_DIGITS];
  __shared__ VOL digit_t b_v[CURVES_BY_BLOCK][NB_DIGITS];
  __shared__ VOL digit_t b_w[CURVES_BY_BLOCK][NB_DIGITS];
  
  unsigned int idx1=blockIdx.x*blockDim.y+threadIdx.y;
  //unsigned int t1=threadIdx.x+1;
  //unsigned int t2=threadIdx.x+NB_DIGITS;
  
  VOL digit_t *t=b_t[threadIdx.y];
  VOL digit_t *u=b_u[threadIdx.y];
  VOL digit_t *v=b_v[threadIdx.y];
  VOL digit_t *w=b_w[threadIdx.y];
  VOL digit_t *temp_r=b_temp_r[threadIdx.y];
  VOL carry_t *cy=b_cy[threadIdx.y];

  //init
  b_cy[threadIdx.y][threadIdx.x]=0; 
  if (threadIdx.x==0)
    b_cy[threadIdx.y][NB_DIGITS]=0; 

  w[threadIdx.x]=x2arg[idx1][threadIdx.x];
  v[threadIdx.x]=z2arg[idx1][threadIdx.x];
  temp_r[threadIdx.x]=xarg[idx1][threadIdx.x];
  u[threadIdx.x]=zarg[idx1][threadIdx.x];

  const digit_t Nthdx = d_Ncst[threadIdx.x]; 
  const digit_t N3thdx = d_3Ncst[threadIdx.x]; 
  const digit_t invN = d_invNcst; 

  Cuda_Add_mod(t, cy, v, w);           /* C=z2+x2 */
  Cuda_Sub_mod(v, cy, w, N3thdx);      /* D=z2-x2 */
  Cuda_Add_mod(w, cy, u, temp_r);      /* A=z+x */
  Cuda_Sub_mod(u, cy, temp_r, N3thdx); /* B=z-x */

  Cuda_Mul_mod(t, cy, t, u, temp_r, Nthdx, invN); /* CB=C*B=(zq+xq)(zp-xp) */
  Cuda_Mul_mod(v, cy, v, w, temp_r, Nthdx, invN); /* DA=D*A=(zq-xq)(zp+xp) */

  Cuda_Square_mod(w, cy, w, temp_r, Nthdx, invN); /* AA=A^2 */
  Cuda_Square_mod(u, cy, u, temp_r, Nthdx, invN); /* BB=B^2 */

  Cuda_Mul_mod(temp_r, cy, u, w, temp_r, Nthdx, invN); /* x2=AA*BB */
  xarg[idx1][threadIdx.x]=temp_r[threadIdx.x];

  Cuda_Sub_mod (w, cy, u, N3thdx); /* C= AA-BB */
  Cuda_Mulint_mod (temp_r, cy, w, idx1 + firstinvd, Nthdx, invN); /* d*C */ 
  Cuda_Add_mod (u, cy, temp_r); /* BB+d*C */
 
  Cuda_Mul_mod (w, cy, w, u, temp_r, Nthdx, invN); /* z2=C*(BB+d*C) */
  zarg[idx1][threadIdx.x]=w[threadIdx.x];
 
  Cuda_Add_mod(w, cy, v, t);       /* DA+CB mod N */
  Cuda_Sub_mod(v, cy, t, N3thdx);  /* DA-CB mod N */

  Cuda_Square_mod(w, cy, w, temp_r, Nthdx, invN); /* (DA+CB)^2 mod N */
  Cuda_Square_mod(v, cy, v, temp_r, Nthdx, invN); /* (DA-CB)^2 mod N */

  /* z0=1 so there is nothing to compute for z0*(DA+CB)^2 */
  Cuda_Dbl_mod(temp_r, v); /* x0=2 x0*(DA-CB)^2 */
  
  x2arg[idx1][threadIdx.x]=w[threadIdx.x];
  z2arg[idx1][threadIdx.x]=temp_r[threadIdx.x];
}

