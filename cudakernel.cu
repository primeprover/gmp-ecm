#include "hip/hip_runtime.h"
/* When compiling the CUDA code, we do not want to include all ecm-impl.h*/
#define _DO_NOT_INCLUDE_ECM_IMPL_H
#include "ecm-gpu.h"
#include <gmp.h>
#include "cudakernel.h"

__constant__ __device__ digit_t d_invNcst;
__device__ biguint_t d_Ncst;
__device__ biguint_t d_3Ncst;
__device__ biguint_t d_Mcst;


#define errCheck(err) cuda_errCheck (err, __FILE__, __LINE__)
#define hipMalloc(d, size) errCheck (hipMalloc (d, size))
#define cudaMemcpyHtoD(d, h, size) errCheck (hipMemcpy ((void *) d, \
                                    (void *) h, size, hipMemcpyHostToDevice))
#define cudaMemcpyDtoH(h, d, size) errCheck (hipMemcpy ((void *) h, \
                                    (void *) d, size, hipMemcpyDeviceToHost))


/******************************/
/* Host code handling the GPU */
/******************************/

inline void cuda_errCheck (hipError_t err, const char *file, const int line)
{
  if( err != hipSuccess ) 
  {
    fprintf(stderr, "%s(%i) : Error cuda : %s.\n",
              file, line, hipGetErrorString( err) );
    exit(EXIT_FAILURE);
  }
}

/* First call to a global function initialize the device */
__global__ void Cuda_Init_Device ()
{
}

extern "C" 
int 
select_and_init_GPU (int device, unsigned int *number_of_curves, int verbose)
{
  hipDeviceProp_t deviceProp;
  hipError_t err;
        
  if (verbose)
      fprintf (stdout, "GPU: compiled for a NVIDIA GPU with compute capability "
                       "%d.%d.\n", ECM_GPU_MAJOR, ECM_GPU_MINOR);

  if (device!=-1)
    {
      if (verbose)
          fprintf (stdout, "GPU: device %d is required.\n", device);

      err = hipSetDevice(device);
      if (err != hipSuccess)
        {
          fprintf (stderr, "GPU: Error: Could not use device %d\n", device);
          fprintf (stderr, "GPU: Error msg: %s\n", hipGetErrorString(err));
          return -1;
        }
    }
  
  err = hipGetDevice (&device);
  if (err != hipSuccess)
    {
      fprintf (stderr, "GPU: Error: no active device.\n");
      fprintf (stderr, "GPU: Error msg: %s\n", hipGetErrorString(err));
      return -1;
    }

  err = hipGetDeviceProperties (&deviceProp, device);
  if (err != hipSuccess)
    {
      fprintf (stderr, "GPU: Error while getting device's properties.\n");
      fprintf (stderr, "GPU: Error msg: %s\n", hipGetErrorString(err));
      return -1;
    }

  int minor = deviceProp.minor;
  int major = deviceProp.major;
  int MPcount = deviceProp.multiProcessorCount;

  if (10 * major + minor < 10 * ECM_GPU_MAJOR + ECM_GPU_MINOR)
    {
      fprintf(stderr, "GPU: Error: device %d have a compute capability of " 
              "%d.%d (required %d.%d).\n", device, major, minor, ECM_GPU_MAJOR,
              ECM_GPU_MINOR);
      return -1;
    }

  if (verbose)
      fprintf (stdout, "GPU: will use device %d: %s, compute capability "
           "%d.%d, %d MPs.\n", device, deviceProp.name, major, minor, MPcount);


  /* number_of_curves should be a multiple of ECM_GPU_CURVES_BY_BLOCK */
  *number_of_curves = 
        (*number_of_curves / ECM_GPU_CURVES_BY_BLOCK) * ECM_GPU_CURVES_BY_BLOCK;
  if (*number_of_curves==0)
    *number_of_curves = MPcount * ECM_GPU_CURVES_BY_MP;

  /* First call to a global function initialize the device */
  errCheck (hipSetDeviceFlags (hipDeviceScheduleYield)); 
  Cuda_Init_Device<<<1, 1>>> ();
  errCheck (hipGetLastError()); 
 
  return 0;
}

extern "C"
float cuda_Main (biguint_t h_N, biguint_t h_3N, biguint_t h_M, digit_t h_invN, 
                    biguint_t *h_xarray, biguint_t *h_zarray, 
                    biguint_t *h_x2array, biguint_t *h_z2array, mpz_t s,
                    unsigned int firstinvd, unsigned int number_of_curves, 
                    int is_verbose_enough) 
{ 
  hipEvent_t start, stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);
  hipEventRecord (start, 0);

  size_t j;
  int i;
  float elltime;
  biguint_t *d_xA, *d_zA, *d_xB, *d_zB;

#define MAXEVENTS 2 
#define DEPTH_EVENT 32
  hipEvent_t event[MAXEVENTS];   // Space for some cuda Event Handles
  long nEventsRecorded = 0;   // Remember how many events are recorded
  long eventrecordix = 0;     // Remember index of next event to record
  long eventsyncix;       // Remember index of oldest recorded event

  size_t array_size = sizeof(biguint_t) * number_of_curves;

  dim3 dimBlock (ECM_GPU_NB_DIGITS, ECM_GPU_CURVES_BY_BLOCK);
  dim3 dimGrid (number_of_curves/ ECM_GPU_CURVES_BY_BLOCK);

  if (is_verbose_enough)
    fprintf(stdout, "Block: %ux%ux%u Grid: %ux%ux%u\n", dimBlock.x, 
                      dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);

  /* Create a pair of events to pace ourselves */
  for (i=0; i<MAXEVENTS; i++)
    errCheck (hipEventCreateWithFlags (&event[i], 
                              hipEventBlockingSync|hipEventDisableTiming));

  hipMalloc (&d_xA, array_size);
  hipMalloc (&d_zA, array_size);
  hipMalloc (&d_xB, array_size);
  hipMalloc (&d_zB, array_size);

  /* Copy into the gpu memory */
  hipMemcpyToSymbol(HIP_SYMBOL(d_invNcst), (void *) &h_invN, sizeof(digit_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_Ncst), (void *) h_N, sizeof(biguint_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_3Ncst), (void *) h_3N, sizeof(biguint_t));
  hipMemcpyToSymbol(HIP_SYMBOL(d_Mcst), (void *) h_M, sizeof(biguint_t));

  cudaMemcpyHtoD (d_xA, h_xarray, array_size);
  cudaMemcpyHtoD (d_zA, h_zarray, array_size);
  cudaMemcpyHtoD (d_xB, h_x2array, array_size);
  cudaMemcpyHtoD (d_zB, h_z2array, array_size);

#ifdef PRINT_REMAINING_ITER
      unsigned int jmod = 100000000;
#endif

  /* Double-and-add loop: it calls the GPU for each bits of s */
  for (j = mpz_sizeinbase (s, 2) - 1; j>0; j-- )
  {
    if (mpz_tstbit (s, j-1) == 1)
      Cuda_Ell_DblAdd<<<dimGrid,dimBlock>>>(d_xB, d_zB, d_xA, d_zA, firstinvd);
    else
      Cuda_Ell_DblAdd<<<dimGrid,dimBlock>>>(d_xA, d_zA, d_xB, d_zB, firstinvd);

    /* Pace entry of events. Less overhead to enter an event every few    */
    /* iterations. But, if you exceed the depth of NVIDIA's kernel queue, */
    /* it will busy-loop!                                                 */
    /* Enter an event every DEPTH_EVENT iteration */
    if (j % DEPTH_EVENT == 0)  
    {
      hipEventRecord(event[eventrecordix]); 
      if (nEventsRecorded == 0)     
        eventsyncix = eventrecordix; 
      nEventsRecorded += 1;          
      eventrecordix = (eventrecordix+1)%MAXEVENTS;  
    }

    if (nEventsRecorded == MAXEVENTS) 
    {
      hipEventSynchronize(event[eventsyncix]);  
      nEventsRecorded -= 1;   
      eventsyncix = (eventsyncix+1)%MAXEVENTS; 
    }

#ifdef PRINT_REMAINING_ITER
    if (j < 100000000) jmod = 10000000;
    if (j < 10000000)  jmod =  1000000;
    if (j < 1000000)   jmod =   100000;
    if (j < 100000)    jmod =    10000;
    if (j % jmod == 0)
      printf("%lu iterations to go\n", j);
#endif
  }

  /* If an error occurs during the kernel calls in the loop */
  errCheck (hipGetLastError()); 

  /* Await for last recorded events */
  while (nEventsRecorded != 0) 
  {
    hipEventSynchronize(event[eventsyncix]); 
    nEventsRecorded -= 1;          
    eventsyncix = (eventsyncix+1)%MAXEVENTS; 
  } 

  /* Get the results back from device memory */
  cudaMemcpyDtoH (h_xarray, d_xA, array_size);
  cudaMemcpyDtoH (h_zarray, d_zA, array_size);

  /* Clean up our events and our stream handle */
  for (i=0; i<MAXEVENTS; i++)
    errCheck (hipEventDestroy(event[i]));


  hipFree ((void *) d_xA);
  hipFree ((void *) d_zA);
  hipFree ((void *) d_xB);
  hipFree ((void *) d_zB);

  hipEventRecord (stop, 0);
  hipEventSynchronize (stop);

  hipEventElapsedTime (&elltime, start, stop);

  errCheck (hipEventDestroy (start));
  errCheck (hipEventDestroy (stop));

  return elltime;
}



/***************/
/* Device code */
/***************/


#define __add_cc(r,a,b) asm ("add.cc.u32 %0, %1, %2;": "=r"(r): "r"(a), "r"(b)) 
#define __addc_cc(r,a,b) asm ("addc.cc.u32 %0, %1, %2;": "=r"(r): "r"(a), "r"(b))
#define __sub_cc(r,a,b) asm ("sub.cc.u32 %0, %1, %2;": "=r"(r): "r"(a), "r"(b)) 

#define __addcy(carry) asm ("addc.s32 %0, 0, 0;": "=r"(carry)) 
#define __addcy2(carry) asm ("addc.cc.s32 %0, %0, 0;": "+r"(carry)) 

#define __subcy(carry) asm ("subc.s32 %0, 0, 0;": "=r"(carry)) 
#define __subcy2(carry) asm ("subc.s32 %0, %0, 0;": "+r"(carry)) 

#define __mul_lo(r,a,b) asm("mul.lo.u32 %0, %1, %2;": "=r"(r): "r"(a),"r"(b)) 
#define __mul_hi(r,a,b) asm("mul.hi.u32 %0, %1, %2;": "=r"(r): "r"(a),"r"(b)) 
#define __mad_lo_cc(r,a,b) asm("mad.lo.cc.u32 %0, %1, %2, %0;":\
                                                      "+r"(r): "r"(a),"r"(b)) 
#define __madc_hi_cc(r,a,b) asm("madc.hi.cc.u32 %0, %1, %2, %0;":\
                                                  "+r"(r):"r"(a),"r"(b)) 


__device__ void Cuda_Fully_Normalize (biguint_t A, dbigint_t cy)
{
  carry_t cytemp;
  unsigned int thm1;

  while(__any(cy[threadIdx.x])!=0)
  {
    thm1 = (threadIdx.x - 1) % ECM_GPU_NB_DIGITS;
    cytemp = cy[thm1];

    __add_cc(A[threadIdx.x], A[threadIdx.x], cytemp);
  
    if (cytemp >= 0)
      __addcy(cy[threadIdx.x]);
    else /* if (cytemp < 0) */
      __subcy(cy[threadIdx.x]);
  }
}

/* Compute Rmod <- A + B */ 
/* Input: 0 <= A, B < 3*N */ 
/* Ouput: 0 <= Rmod < 6*N */ 
__device__ void Cuda_Add_mod
(biguint_t Rmod, dbigint_t cy, const biguint_t A, const biguint_t B)
{
  unsigned int thp1 = (threadIdx.x + 1) % ECM_GPU_NB_DIGITS;
  __add_cc (Rmod[threadIdx.x], A[threadIdx.x], B[threadIdx.x]);
  __addcy2(Rmod[thp1]); 
  __addcy (cy[thp1]);
  Cuda_Fully_Normalize (Rmod, cy); 
}

/* Compute Rmod <- Rmod + B */ 
/* Input: 0 <= Rmod, B < 3*N */ 
/* (except when it follows Cuda_Mulint_mod, 0 <= Rmod < 3*N, 0 < B < 7*N ) */ 
/* Ouput: 0 <= Rmod < 6*N */ 
/* (except when it follows Cuda_Mulint_mod, 0 <= Rmod < 10*N) */ 
__device__ void Cuda_Add_mod
(biguint_t Rmod, dbigint_t cy, const biguint_t A)
{
  unsigned int thp1 = (threadIdx.x + 1) % ECM_GPU_NB_DIGITS;
  __add_cc (Rmod[threadIdx.x], Rmod[threadIdx.x], A[threadIdx.x]);
  //__addcy (cy[threadIdx.x]);
  __addcy2(Rmod[thp1]); 
  __addcy (cy[thp1]);
  Cuda_Fully_Normalize (Rmod, cy);
}

/* Compute Rmod <- Rmod - B */ 
/* Input: 0 <= Rmod, B < 3*N */ 
/* Ouput: 0 <= Rmod < 6*N */ 
__device__ void Cuda_Sub_mod 
(biguint_t Rmod, dbigint_t cy, const biguint_t B, const digit_t N3thdx)
{
  digit_t reg_Rmod = Rmod[threadIdx.x];
  carry_t reg_cy = 0; 
  
  __add_cc (reg_Rmod, reg_Rmod, N3thdx);
  __addcy (reg_cy);
  __sub_cc (reg_Rmod, reg_Rmod, B[threadIdx.x]);
  __subcy2 (reg_cy);

  Rmod[threadIdx.x] = reg_Rmod;
  cy[threadIdx.x] = reg_cy;
  Cuda_Fully_Normalize (Rmod, cy); 
}

/* Perform one step of REDC */ 
__device__ void Cuda_Mulmod_step
(dbiguint_t r, dbigint_t cy, digit_t a, digit_t b, const digit_t Nthdx,
 const digit_t invN)
{
  digit_t t;
  digit_t reg_hi = 0;
  unsigned int thp1= (threadIdx.x + 1) % ECM_GPU_NB_DIGITS;
  carry_t reg_cy = cy[thp1];

  __mad_lo_cc(r[threadIdx.x],a,b);
  __madc_hi_cc(reg_hi,a,b);
  __addcy2(reg_cy);

  __mul_lo(t, invN, r[0]);
  __mad_lo_cc(r[threadIdx.x],t,Nthdx);
  __madc_hi_cc(reg_hi,t,Nthdx);
  __addcy2(reg_cy);

  /* make one round of normalize + a right shift at the same time */
  __add_cc(r[threadIdx.x],r[thp1],reg_hi);
  __addc_cc(r[thp1],r[thp1],reg_cy);
  __addcy(cy[thp1]); 
}

/* Compute r <- 2*a */ 
/* Input: 0 <= a < 3*N */ 
/* Ouput: 0 <= r < 3*N */ 
__device__ void Cuda_Dbl_mod
(biguint_t r, biguint_t a)
{
  unsigned int thp1= (threadIdx.x + 1) % ECM_GPU_NB_DIGITS;
  asm ("add.cc.u32 %0, %1, %1;" : "=r"(r[threadIdx.x]) : "r"(a[threadIdx.x]));
  __addcy2(r[thp1]);
}


/* Compute r <- A*b */ 
/* Input: 0 < b < 2^SIZE_DIGIT, 0 <= A < 6*N */ 
/* Ouput: 0 <= r < 7*N */ 
__device__ void Cuda_Mulint_mod
(dbiguint_t r, dbigint_t cy, biguint_t A, digit_t b, const digit_t Nthdx,
 const digit_t invN)
{
  digit_t t;
  digit_t reg_hi;
  unsigned int thp1= (threadIdx.x + 1) % ECM_GPU_NB_DIGITS;
  digit_t reg_A = A[threadIdx.x];
  carry_t reg_cy;

  __mul_lo(r[threadIdx.x],reg_A,b);
  __mul_hi(reg_hi,reg_A,b);

  __mul_lo(t, invN, r[0]);
  __mad_lo_cc(r[threadIdx.x],t,Nthdx);
  __madc_hi_cc(reg_hi,t,Nthdx);
  __addcy(reg_cy);

  /* make one round of normalize + a right shift at the same time */
  __add_cc(r[threadIdx.x],r[thp1],reg_hi);
  __addc_cc(r[thp1],r[thp1],reg_cy);
  __addcy(cy[thp1]); 

  Cuda_Fully_Normalize(r,cy); 
}

/* Compute r <- A*B */ 
/* Input: 0 <= A, B < 6*N */
/* (except when it follows Cuda_Mulint_mod, 0 <= A < 6*N, 0 < B < 10*N ) */ 
/* Ouput: 0 <= r < 3*N */ 
__device__ void Cuda_Mul_mod 
(biguint_t mul, dbigint_t cy, const biguint_t A, const biguint_t B, dbiguint_t r,
 const digit_t Nthdx, const digit_t invN)
{

  int i;
  digit_t temp=A[threadIdx.x];

  r[threadIdx.x]=0;
  
  for (i=0; i<ECM_GPU_NB_DIGITS; i++)
    Cuda_Mulmod_step (r, cy, temp, B[i], Nthdx, invN);

  
  Cuda_Fully_Normalize (r, cy);
  mul[threadIdx.x]=r[threadIdx.x];
}

__device__ void Cuda_Square_mod 
(biguint_t mul, dbigint_t cy, const biguint_t A, dbiguint_t r, 
 const digit_t Nthdx, const digit_t invN)
{
  Cuda_Mul_mod (mul, cy, A, A, r, Nthdx, invN);
}

/* 
  Compute silmutaneously:
  (xarg : zarg ) <- [2](xarg : zarg) 
  (xarg2 : zarg2 ) <- (xarg : zarg) + (xarg2 : zarg2) 
*/
__global__ void 
Cuda_Ell_DblAdd (biguint_t *xAarg, biguint_t *zAarg, biguint_t *xBarg, 
                                       biguint_t *zBarg, unsigned int firstinvd)
{
  __shared__ VOL digit_t b_temp_r[ECM_GPU_CURVES_BY_BLOCK][ECM_GPU_NB_DIGITS];
  __shared__ VOL carry_t b_cy[ECM_GPU_CURVES_BY_BLOCK][ECM_GPU_NB_DIGITS]; 

  __shared__ VOL digit_t b_t[ECM_GPU_CURVES_BY_BLOCK][ECM_GPU_NB_DIGITS];
  __shared__ VOL digit_t b_u[ECM_GPU_CURVES_BY_BLOCK][ECM_GPU_NB_DIGITS];
  __shared__ VOL digit_t b_v[ECM_GPU_CURVES_BY_BLOCK][ECM_GPU_NB_DIGITS];
  __shared__ VOL digit_t b_w[ECM_GPU_CURVES_BY_BLOCK][ECM_GPU_NB_DIGITS];
  
  VOL digit_t *t=b_t[threadIdx.y];
  VOL digit_t *u=b_u[threadIdx.y];
  VOL digit_t *v=b_v[threadIdx.y];
  VOL digit_t *w=b_w[threadIdx.y];
  VOL digit_t *temp_r=b_temp_r[threadIdx.y];
  VOL carry_t *cy=b_cy[threadIdx.y];

  /* Init of shared variables */
  const unsigned int idx1=blockIdx.x*blockDim.y+threadIdx.y;
  //unsigned int t1=threadIdx.x+1;
  cy[threadIdx.x]=0; 

  w[threadIdx.x]=xBarg[idx1][threadIdx.x];
  v[threadIdx.x]=zBarg[idx1][threadIdx.x];
  temp_r[threadIdx.x]=xAarg[idx1][threadIdx.x];
  u[threadIdx.x]=zAarg[idx1][threadIdx.x];

  const digit_t Nthdx = d_Ncst[threadIdx.x]; 
  const digit_t N3thdx = d_3Ncst[threadIdx.x]; 
  const digit_t invN = d_invNcst; 

  Cuda_Add_mod(t, cy, v, w);           /* C=zB+xB */
  Cuda_Sub_mod(v, cy, w, N3thdx);      /* D=zB-xB */
  Cuda_Add_mod(w, cy, u, temp_r);      /* A=zA+xA */
  Cuda_Sub_mod(u, cy, temp_r, N3thdx); /* B=zA-xA */

  Cuda_Mul_mod(t, cy, t, u, temp_r, Nthdx, invN); /* CB=C*B=(zB+xB)(zA-xA) */
  Cuda_Mul_mod(v, cy, v, w, temp_r, Nthdx, invN); /* DA=D*A=(zB-xB)(zA+xA) */

  Cuda_Square_mod(w, cy, w, temp_r, Nthdx, invN); /* AA=A^2 */
  Cuda_Square_mod(u, cy, u, temp_r, Nthdx, invN); /* BB=B^2 */

  Cuda_Mul_mod(temp_r, cy, u, w, temp_r, Nthdx, invN); /* AA*BB */
  xAarg[idx1][threadIdx.x]=temp_r[threadIdx.x];

  Cuda_Sub_mod (w, cy, u, N3thdx); /* K= AA-BB */
  Cuda_Mulint_mod (temp_r, cy, w, idx1 + firstinvd, Nthdx, invN); /* d*K */ 
  Cuda_Add_mod (u, cy, temp_r); /* BB+d*K */
 
  Cuda_Mul_mod (w, cy, w, u, temp_r, Nthdx, invN); /* K*(BB+d*K) */
  zAarg[idx1][threadIdx.x]=w[threadIdx.x];
 
  Cuda_Add_mod(w, cy, v, t);       /* DA+CB mod N */
  Cuda_Sub_mod(v, cy, t, N3thdx);  /* DA-CB mod N */

  Cuda_Square_mod(w, cy, w, temp_r, Nthdx, invN); /* (DA+CB)^2 mod N */
  Cuda_Square_mod(v, cy, v, temp_r, Nthdx, invN); /* (DA-CB)^2 mod N */

  /* z0=1 so there is nothing to compute for z0*(DA+CB)^2 */
  Cuda_Dbl_mod(temp_r, v); /* x0=2 x0*(DA-CB)^2 */
  
  xBarg[idx1][threadIdx.x]=w[threadIdx.x];
  zBarg[idx1][threadIdx.x]=temp_r[threadIdx.x];
}
